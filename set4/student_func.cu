#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

/* Red Eye Removal
   ===============

   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */
const unsigned int MAX_THREADS = 1024;
const unsigned int NUM_BITS    = 1;
const unsigned int NUM_BINS    = 1 << NUM_BITS;

//HELPERS----------------------------------------------------------------------

void displayCudaBufferWindow(const unsigned int* const d_buf,
                             const size_t              numElems,
                             const size_t              from,
                             const size_t              to) {
    unsigned int *buf = new unsigned int[numElems];
    checkCudaErrors(hipMemcpy(buf,  d_buf,  sizeof(unsigned int) * numElems, hipMemcpyDeviceToHost));
    for (int i = from ; i < to; ++i) {
        std::cout << std::hex << buf[i] << " " << std::endl;
    }
    std::cout << std::endl;

    delete[] buf;
}

void displayReducedArray(const unsigned int* const d_buf,
                         const size_t              size) {
    int ssize = size / MAX_THREADS;
    unsigned int *buf = new unsigned int[size];
    checkCudaErrors(hipMemcpy(buf,  d_buf,  sizeof(unsigned int) * size, hipMemcpyDeviceToHost));
    if (ssize > 1) {
        int interval = size / ssize;

        std::cout << std::hex << "REDUCED" << std::endl;
        for (int myId = 0; myId < ssize; ++myId) {
            std::cout << std::dec << buf[myId * interval + interval - 1] << " " << std::endl;
        }
    }
}

void displayCudaBuffer(const unsigned int* const d_buf,
                       const size_t              numElems) {
  displayCudaBufferWindow(d_buf, numElems, 0, numElems);
}

unsigned int displayCudaBufferMax(const unsigned int* const d_buf,
                                  const size_t              numElems) {
  unsigned int *buf = new unsigned int[numElems];
  checkCudaErrors(hipMemcpy(buf,  d_buf,  sizeof(unsigned int) * numElems, hipMemcpyDeviceToHost));
  unsigned int max = buf[0];
  unsigned int idx = 0;
  int checksum = 0;
  for (int i = 0 ; i < numElems; ++i) {
      checksum += buf[i];
      if (max <= buf[i]) {
          max = buf[i];
          idx = i;
      }
  }
  std::cout << "checksum " << std::dec << checksum << std::endl;
  std::cout << "max " << std::dec << max << " idx " << idx << std::endl;
  int begin = idx - 10;
  if (begin < 0) begin = 0;

  int lastIndex = idx + 10;
  if (lastIndex > numElems) {
      lastIndex = idx + 1;
  }

  displayCudaBufferWindow(d_buf, numElems, begin, lastIndex);
  std::cout << "last " << std::endl;
  if (numElems > 50) {
      displayCudaBufferWindow(d_buf, numElems, numElems - 50, numElems);
  } else {
      displayCudaBufferWindow(d_buf, numElems, 0,             numElems);
  }

  delete[] buf;
  return max;
}

__device__ unsigned int myMin(const unsigned int a,
                              const unsigned int b) {
    if (a < b) return a;
    return b;
}

__global__ void fill10 (      unsigned int* const d_dst,
                        const unsigned int numElems) {
    unsigned int tid  = threadIdx.x;
    unsigned int myId = tid + blockDim.x * blockIdx.x;
    if (myId >= numElems) {
        return;
    }
    d_dst[myId] = (myId + 1) % 2;
}

//ALGORITHMS-------------------------------------------------------------------

__global__ void histogram(const unsigned int* const d_in,
                                unsigned int* const d_res,
                          const unsigned int        mask,
                          const unsigned int        i,
                          const size_t              numElems) {
    unsigned int tid  = threadIdx.x;
    unsigned int myId = tid + blockDim.x * blockIdx.x;
    if (myId < NUM_BINS) { //������� ������ ����������
       d_res[myId] = 0;
    }
    if (myId >= numElems) {
        return;
    }
    unsigned int binId = (d_in[myId] & mask) >> i;

    __syncthreads();
    atomicAdd(&(d_res[binId]), 1);
}

__device__ void scanReduceForBlock(      unsigned int* const d_res,
                                   const size_t              maxDisplacement,
                                   const unsigned int        size,
                                   const unsigned int        myId) {
    unsigned int nextId;
    unsigned int prevValue;
    unsigned int nextValue;

    for (unsigned int s = 1; s <= maxDisplacement / 2; s *= 2) {

        __syncthreads();
        prevValue = d_res[myId];
        nextId    = myId + s;
        nextValue = nextId < size ? d_res[nextId] : 0;

        __syncthreads();
        if (((nextId + 1) % (s * 2)) == 0 && (nextId < size)) {
            d_res[nextId] = prevValue + nextValue;
        }
    }
}

__device__  void scanDownStepDevice(      unsigned int* const d_res,
                                    const unsigned int        initialS,
                                    const unsigned int        myId) {
    int prevId;
    unsigned int prevValue;
    unsigned int myValue;

    for (unsigned int s = initialS; s >= 2; s /= 2) {

        __syncthreads();
        prevId    = myId - s / 2;
        prevValue = (prevId >= 0) ? d_res[prevId] : 0;
        myValue   =                 d_res[myId];

        __syncthreads();
        if (((myId + 1) % s) == 0 && prevId >= 0) {
            d_res[prevId] = myValue;
            d_res[myId]   = myValue + prevValue;
        }
    }
}

__global__  void compact(const unsigned int* const d_in,
                               unsigned int* const d_res,
                         const size_t              size,
                         const int                 ssize) {
    unsigned int tid  = threadIdx.x;
    unsigned const int myId = tid + (blockDim.x) * blockIdx.x;
    if (myId >=size) {
        return;
    }
    unsigned int interval = size/ ssize;
    const unsigned int reducedId = myId / interval;
    int myCurrentIndex = reducedId * interval + interval - 1;
    if (myId > 0 && myId % myCurrentIndex == 0) { //��������� ������ ������ ������ �����
        d_res[reducedId] = d_res[myCurrentIndex];
    }
}

__global__  void enlarge(const unsigned int* const d_in,
                               unsigned int* const d_res,
                         const size_t              size,
                         const int                 ssize) {
    unsigned int tid  = threadIdx.x;
    unsigned const int myId = tid + (blockDim.x) * blockIdx.x;
    if (myId >=size) {
        return;
    }
    __syncthreads();
    unsigned int interval = size/ ssize;
    const unsigned int reducedId = myId / interval;
    int myCurrentIndex = reducedId * interval + interval - 1;
    if (myId > 0 && myId % myCurrentIndex == 0) { //��������� ������ ������ ������ �����
        d_res[myCurrentIndex] = d_res[reducedId];
    }
}

__global__  void blellochBlockScan(const unsigned int* const d_in,
                                         unsigned int* const d_res,
                                   const size_t              size) {
    unsigned int tid  = threadIdx.x;
    unsigned const int myId = tid + (blockDim.x) * blockIdx.x;
    if (myId >=size) {
        return;
    }
    scanReduceForBlock(d_res, size, size, myId);

    __syncthreads();
    d_res[size-1] = 0;

    __syncthreads();
     scanDownStepDevice(d_res, size, myId);
}

__global__  void blellochBigScan(const unsigned int* const d_in,
                                       unsigned int* const d_res,
                                 const size_t              size) {
    extern __shared__ unsigned int sdata[];
    unsigned int tid  = threadIdx.x;
    unsigned const int myId = tid + (blockDim.x) * blockIdx.x;
    if (myId >=size) {
        return;
    }
    d_res[myId] = d_in[myId];
    scanReduceForBlock(d_res, myMin(MAX_THREADS, size), size, myId);
}

__global__  void blellochBigScanDownstep(      unsigned int* const d_res,
                                         const size_t              size) {
    unsigned int tid = threadIdx.x;
    unsigned int myId = tid + (blockDim.x) * blockIdx.x;
    if (myId >= size) {
        return;
    }
    unsigned int initialS = myMin(MAX_THREADS, size);
    d_res[size - 1] = 0;
    scanDownStepDevice(d_res, initialS, myId);
}


/**
d_binScan - ��� ������� �������� �������,
�������� ��������, ���� ����� �������� ���������, � ������ ���� �� ����� � ��� �������
d_vals_dst - �������� �������� ��� ������� id ��� ����������� �������
d_vals_dst ����� ����� ��������� ���������
**/
__global__ void gather(const unsigned int* const d_vals_src,
                       const unsigned int* const d_pos_src,
                       const unsigned int* const d_new_index_src,
                             unsigned int* const d_vals_dst,
                             unsigned int* const d_pos_dst,
                       const unsigned int        numElems) {
    unsigned int tid  = threadIdx.x;
    unsigned int myId = tid + (blockDim.x) * blockIdx.x;

    if (myId >= numElems) {
        return;
    }

    __syncthreads();
    unsigned int newIndex = d_new_index_src[myId];

    __syncthreads();
    d_vals_dst[newIndex] = d_vals_src[myId];
    d_pos_dst[newIndex]  = d_pos_src[myId];
}

/**
d_binScan - ��� ������� �������� �������,
�������� ��������, ���� ����� �������� ���������, � ������ ���� �� ����� � ��� �������
d_disp_src - �������� �������� ��� ������� id ��� ����������� �������
**/
__global__ void getNewIndexes(const unsigned int* const d_vals_src,
                              const unsigned int* const d_disp_src,
                              const unsigned int* const d_binScan,
                                    unsigned int* const d_new_index_dst,
                              const unsigned int        mask,
                              const unsigned int        i,
                              const unsigned int        numElems) {
    unsigned int tid  = threadIdx.x;
    unsigned int myId = tid + (blockDim.x) * blockIdx.x;

    if (myId >= numElems) {
        return;
    }
    unsigned int myIdOffset = d_disp_src[myId]; //� ��� ���� ��, ��� ������ �� ���������� ��������,
    //������ � ��������� ����������
    unsigned int binId = (d_vals_src[myId] & mask) >> i;

    __syncthreads();
    unsigned int offset   = d_binScan[binId];
    unsigned int newIndex = offset + myIdOffset;
    d_new_index_dst[myId] = newIndex;
}

__global__ void mapToBin(const unsigned int* const d_vals_src,
                               unsigned int* const d_vals_dst,
                         const unsigned int        mask,
                         const unsigned int        i,
                         const unsigned int        mappedBean,
                         const unsigned int        numElems) {
    unsigned int tid  = threadIdx.x;
    unsigned int myId = tid + (blockDim.x) * blockIdx.x;

    if (myId >= numElems) {
        return;
    }
    unsigned int beanId = (d_vals_src[myId] & mask) >> i;
    d_vals_dst[myId] = (beanId == mappedBean) ? 1 : 0;
}

__global__ void resetMapToBin(const unsigned int* const d_vals_src,
                                    unsigned int* const d_vals_dst,
                              const unsigned int        mask,
                              const unsigned int        i,
                              const unsigned int        mappedBean,
                              const unsigned int        numElems) {
    unsigned int tid  = threadIdx.x;
    unsigned int myId = tid + (blockDim.x) * blockIdx.x;

    if (myId >= numElems) {
        return;
    }
    int beanId = (d_vals_src[myId] & mask) >> i;
    if (beanId != mappedBean) {
        d_vals_dst[myId] = 0;
    }
}

__global__ void clear(      unsigned int* const d_vals_dst,
                      const unsigned int        numElems) {
    unsigned int tid  = threadIdx.x;
    unsigned int myId = tid + (blockDim.x) * blockIdx.x;

    if (myId >= numElems) {
        return;
    }
    d_vals_dst[myId] = 0;
}

__global__ void copy(const unsigned int* const d_src,
                           unsigned int* const d_dst,
                     const unsigned int        numElems) {
    unsigned int tid  = threadIdx.x;
    unsigned int myId = tid + blockDim.x * blockIdx.x;
    if (myId >= numElems) {
        return;
    }
    d_dst[myId] = d_src[myId];
}

__global__ void sum(const unsigned int* const d_src,
                          unsigned int* const d_dst,
                    const unsigned int numElems) {
    unsigned int tid  = threadIdx.x;
    unsigned int myId = tid + blockDim.x * blockIdx.x;
    if (myId >= numElems) {
        return;
    }
    d_dst[myId] = d_src[myId] + d_dst[myId];
}

unsigned int getNearest(unsigned int const number) {
    unsigned int result = 1;
    while( result < number ) {
        result <<= 1;
    }
    return result;
}

//MAIN--------------------------------------------------------------------

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               size_t              numElems)
{
  unsigned int* d_binScan;
  unsigned int* d_binHistogram;
  unsigned int* d_temp;
  unsigned int* d_temp1;
  unsigned int* d_iv = d_inputVals;
  unsigned int* d_ip = d_inputPos;
  unsigned int* d_ov = d_outputVals;
  unsigned int* d_op = d_outputPos;
  unsigned int* sdata;

  numElems = 16383;//32;//16;//18000;
  int elemstoDisplay = 16;

  int alignedBuferElems = getNearest(numElems);
  int ssize             = alignedBuferElems / MAX_THREADS; //������� ��������� ����� � ����������� �������

  checkCudaErrors(hipMalloc((void **) &d_binScan,      NUM_BINS          * sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((void **) &d_binHistogram, NUM_BINS          * sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((void **) &d_temp,         alignedBuferElems * sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((void **) &d_temp1,        alignedBuferElems * sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((void **) &sdata,          ssize             * sizeof(unsigned int)));

  const unsigned int numBlocksForAligned  = (alignedBuferElems + MAX_THREADS - 1) / MAX_THREADS;
  const unsigned int numBlocksForElements = (numElems          + MAX_THREADS - 1) / MAX_THREADS;

  fill10 <<<numBlocksForElements, MAX_THREADS>>> (d_iv, numElems);

  std::cout << "numElems " << numElems << std::endl;
  std::cout << "NUM_BINS " << NUM_BINS << std::endl;

  std::cout << "d_inputVals " << std::endl;
  displayCudaBuffer(d_inputVals, elemstoDisplay);



  //a simple radix sort - only guaranteed to work for NUM_BITS that are multiples of 2
  for (unsigned int i = 0; i < 8 * sizeof(unsigned int); i += NUM_BITS) {
      unsigned int mask = (NUM_BINS - 1) << i;

      clear <<<numBlocksForElements, MAX_THREADS>>> (d_ov,numElems);

      for (unsigned int j = 0; j < NUM_BINS; ++j) {
          //checkCudaErrors(hipMemset(d_temp, 0,  sizeof(unsigned int) * alignedBuferElems));
          clear <<<numBlocksForAligned, MAX_THREADS>>> (d_temp,  alignedBuferElems);
          clear <<<numBlocksForAligned, MAX_THREADS>>> (d_temp1, alignedBuferElems);

          mapToBin <<<numBlocksForElements, MAX_THREADS>>> (d_iv, d_temp, mask, i, j, numElems);
          //std::cout << "mapToBin" << j << " " <<  mask << " " << i << std::endl;
          //displayCudaBuffer(d_temp, elemstoDisplay);
          //std::cout << "DEEP " << std::endl;
          //displayCudaBufferWindow(d_temp, numElems, 2000, 2010);
          //displayCudaBufferMax(d_temp, alignedBuferElems);

          blellochBigScan <<<numBlocksForAligned, MAX_THREADS>>>
                       (d_temp, d_temp1, alignedBuferElems);
          if (ssize > 1) {
              compact <<<numBlocksForAligned, MAX_THREADS>>>
                       (d_temp1, sdata, alignedBuferElems, ssize);
              blellochBlockScan <<<1, ssize>>> (sdata, sdata, ssize);
              enlarge <<<numBlocksForAligned, MAX_THREADS>>>
                       (sdata, d_temp1, alignedBuferElems, ssize);
          }
          blellochBigScanDownstep <<<numBlocksForAligned, MAX_THREADS>>>
                       (d_temp1, alignedBuferElems);

          std::cout << "scan " << std::endl;
          displayCudaBuffer(d_temp1,   elemstoDisplay);
          displayReducedArray(d_temp1, alignedBuferElems);
          unsigned int max = displayCudaBufferMax(d_temp1, alignedBuferElems);

          resetMapToBin <<<numBlocksForElements, MAX_THREADS>>>
                        (d_iv, d_temp1, mask, i, j, numElems);
          std::cout << "resetMapToBin " << std::endl;
          displayCudaBuffer(d_temp1, elemstoDisplay);
          displayCudaBufferMax(d_temp1, numElems);

          sum <<<numBlocksForElements, MAX_THREADS>>>
              (d_temp1,d_ov,numElems);
          std::cout << "sum " << std::endl;
          displayCudaBufferMax(d_ov, numElems);
          displayCudaBuffer(d_ov, elemstoDisplay);
      }

      histogram <<<numBlocksForElements, MAX_THREADS>>> (d_iv, d_binHistogram, mask, i, numElems);
      //histogram<<<1, numElems>>>(d_iv, d_binHistogram, mask, i, numElems, NUM_BINS);
      std::cout << "d_binHistogram " << std::endl;
      displayCudaBuffer(d_binHistogram, NUM_BINS);

      //perform exclusive prefix sum (scan) on binHistogram to get starting
      //location for each bin
      blellochBlockScan <<<1, NUM_BINS>>>
                           (d_binHistogram, d_binScan, NUM_BINS);
      std::cout << "d_binScan " << std::endl;
      displayCudaBuffer(d_binScan, NUM_BINS);

      //Gather everything into the correct location
      //need to move vals and positions
      unsigned int* d_disp_src  = d_ov;
      unsigned int* d_new_index = d_op;
      displayCudaBufferMax(d_disp_src, numElems);
      getNewIndexes <<<numBlocksForElements, MAX_THREADS>>>
                    (d_iv, d_disp_src, d_binScan, d_new_index, mask, i, numElems);
      std::cout << "after getNewIndexes " << std::endl;
      displayCudaBuffer(d_new_index, elemstoDisplay);
      gather <<<numBlocksForElements, MAX_THREADS>>>
             (d_iv, d_ip, d_new_index, d_ov, d_op, numElems);
      std::cout << "after gather " << std::endl;
      displayCudaBuffer(d_ov, elemstoDisplay);

      //swap the buffers (pointers only)
      std::swap(d_ov, d_iv);
      std::swap(d_op, d_ip);
  }

  //we did an even number of iterations, need to copy from input buffer into output
  copy <<<numBlocksForElements, MAX_THREADS>>> (d_iv, d_ov, numElems);
  copy <<<numBlocksForElements, MAX_THREADS>>> (d_ip, d_op, numElems);

  std::cout << "d_outputVals " << std::endl;
  displayCudaBuffer(d_outputVals, elemstoDisplay);
  displayCudaBufferMax(d_outputVals, numElems);
  std::cout << "d_inputVals " << std::endl;
  displayCudaBuffer(d_inputVals, elemstoDisplay);
  displayCudaBufferMax(d_inputVals, numElems);

  checkCudaErrors(hipFree(d_binScan));
  checkCudaErrors(hipFree(d_binHistogram));
  checkCudaErrors(hipFree(d_temp));
  checkCudaErrors(hipFree(d_temp1));
  checkCudaErrors(hipFree(sdata));
}



